#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <algorithm>
#include "sarray.h"
#include "qsort.h"

#define CUDA_SAFE_CALL(x) x
using namespace std;

__device__  int g_pivotIndex;

void print(int *array, int n)
{
    for(int i=0; i < n; i++)
        cout << array[i] << endl;
}

void print_gene_array(int *array, int n)
{
    for(int i=0;i<n;i++){
        printf("%d - %s\n", array[i], cpu_genome+array[i]);
    }
}

int setup( int num , char* filename ) 
{
    cpu_genome = (char *) malloc(sizeof(char)*(num+1));
    read_genome2(filename, cpu_genome, num);
    return (strlen(cpu_genome));        
}

void read_genome2(char *filename, char *buffer, int num)
{
    FILE *fh;
    fh = fopen(filename, "r");
    fread(buffer, 1, num, fh);
    buffer[num] = '\0';
    fclose(fh);
}

__device__ int strcmp_cuda(char *source, char *dest)
{
	int i;
	for (i = 0; source[i] == dest[i]; i++)
		if (source[i] == '\0')
			return 0;
	return source[i] - dest[i];
}

__global__ void block_scan_write_up( int *g_idata, int block_offset, int block_size, int start, int end, int n)
{
    int tid = (int) (threadIdx.x + blockDim.x * threadIdx.y + \
							( blockIdx.x * blockDim.x * blockDim.y ) \
							+ ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x)); 

    int prev_block_offset = block_offset/block_size;

    if((((tid+1)*prev_block_offset) - 1) >=start && (((tid+1)*prev_block_offset) - 1)<=end)
    {
	        
	    int prev_block_index = (((tid+1)*prev_block_offset) - 1);
	    
	    int x = prev_block_index/block_offset;

	    if((prev_block_index+1) % block_offset != 0 && x > 0)
	    {
	    	g_idata[prev_block_index] += g_idata[(x)*block_offset - 1];    	
	    }    	
    }
}

__global__ void block_scan( int *g_idata, int block_offset, int block_size, int start, int end, int n) 
{ 
    extern __shared__ int temp[]; // allocated on invocation 

    int tid = (int) (threadIdx.x + blockDim.x * threadIdx.y + \
							( blockIdx.x * blockDim.x * blockDim.y ) \
							+ ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x)); 

    int thid = threadIdx.x; 
	int pout = 0, pin = 1; 
	int n1 = block_size;
	temp[thid] = 0;
	temp[n1+thid] = 0;

    // load input into shared memory.  
    // This is exclusive scan, so shift right by one and set first elt to 0 
	if( (((tid+1)*block_offset) - 1) >= start && (((tid+1)*block_offset) - 1) <= end)
	{
	    temp[pout*n1 + thid] = g_idata[((tid+1)*block_offset) - 1];  
	    temp[pin*n1 + thid] = g_idata[((tid+1)*block_offset) - 1]; 
	}
	else
	{
	    temp[pout*n1 + thid] = 0; 
	    temp[pin*n1 + thid] = 0; 
	}

    __syncthreads(); 

    for (int offset = 1; offset < n1; offset *= 2) 
    { 
        pout = 1 - pout; // swap double buffer indices 
        pin  = 1 - pin; 

        if (thid >= offset) 
            temp[pout*n1+thid] = temp[pin*n1+thid] + temp[pin*n1+thid - offset]; 
   		else 
  			temp[pout*n1+thid] = temp[pin*n1+thid]; 

        __syncthreads(); 
    }

	if( (((tid+1)*block_offset) - 1) >= start && (((tid+1)*block_offset) - 1) <= end)
		g_idata[((tid+1)*block_offset) - 1] = temp[pout*n1+thid];
}

void prefixCompute( int *gpu_prefix_arr, dim3 blockGridRows, dim3 threadBlockRows, 
                    int block_size, int start, int end, int prefixesCount)
{
	int sharedMemory = 2*block_size*sizeof(int);
	
	int block_offset = 0;
	int l = log2((float)prefixesCount)/log2((float)block_size);

	for(int i=0; i<=l; i++)
	{
		block_offset = pow((float)block_size, i);
		block_scan <<<  blockGridRows, 
                        threadBlockRows, 
                        sharedMemory >>> (  gpu_prefix_arr, block_offset, 
                                            block_size, start, end, 
                                            prefixesCount);
		hipDeviceSynchronize();
	}
	for(int i=l; i > 0; i--)
	{
		block_offset = pow((float)block_size, i);	
		block_scan_write_up <<< blockGridRows, 
                                threadBlockRows, 
                                sharedMemory>>>(gpu_prefix_arr, block_offset, block_size, 
                                                start, end, prefixesCount);
		hipDeviceSynchronize();
	}

}

__device__ int get_pivot_index(int start, int end)
{
    /*
     * TODO : Select the Pivot Index using either some operations using
     * property of DNA string or Sampling or any form of randomization
     * which will split the recursive call between 1/4 to 3/4 for 
     * efficient running of Quick Sort. For now selecting random/mid-point value
     * as pivot.
     *
     */

    return start + (end - start)/2;
}

__global__ void quickSortGPU(   int *sh_gpu_suf_arr, int *sh_gpu_suf_arr_copy, int *sh_gpu_aux_arr, 
                                int *start_ind_arr, int *end_ind_arr, int *start_ind_arr_copy, 
                                int *end_ind_arr_copy, char* sh_gpu_genome,  int block_size, int suff_size)
{
    extern __shared__ int shared[];
    int *lt_pivot = &shared[0];

    int tid = (int) (threadIdx.x + blockDim.x * threadIdx.y + \
                            ( blockIdx.x * blockDim.x * blockDim.y ) \
                            + ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x));

    int pivotIndex;
    if(tid < suff_size)
    {
        start_ind_arr[tid] = start_ind_arr_copy[tid];
        end_ind_arr[tid] = end_ind_arr_copy[tid];
        sh_gpu_suf_arr_copy[tid] = sh_gpu_suf_arr[tid];

        if(start_ind_arr[tid] != -1)
        {
            lt_pivot[threadIdx.x] = 0;
            pivotIndex = get_pivot_index(start_ind_arr[tid], end_ind_arr[tid]);

            if(tid != pivotIndex)
            {
                if(strcmp_cuda(sh_gpu_genome + sh_gpu_suf_arr[tid] ,sh_gpu_genome + sh_gpu_suf_arr[pivotIndex] ) < 0)
                {
                    lt_pivot[threadIdx.x] = 1;
                }
                else
                {
                    lt_pivot[threadIdx.x] = 0;
                }
            }   
        }
        else
        {
            lt_pivot[threadIdx.x] = 0;
        }       
        __syncthreads(); // Barrier - all threads must finish comparing suffixes 

        sh_gpu_aux_arr[tid] = lt_pivot[threadIdx.x];
    }
}

__device__ int terminate_t = 1;

__global__ void write_pivot(int *sh_gpu_suf_arr, int *sh_gpu_suf_arr_copy, int *sh_gpu_aux_arr, 
                            int *start_ind_arr, int *end_ind_arr, int *start_ind_arr_copy, 
                            int *end_ind_arr_copy, int n)
{
    int tid = (int) (threadIdx.x + blockDim.x * threadIdx.y + \
                            ( blockIdx.x * blockDim.x * blockDim.y ) \
                            + ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x));

    int pivotIndex, pivotIndexPosition;
    pivotIndex = get_pivot_index(start_ind_arr[tid], end_ind_arr[tid]);
    if(tid == 0)
    {
        terminate_t = 0;
    }
    if(tid < n)
    {

        pivotIndexPosition = start_ind_arr[tid] + sh_gpu_aux_arr[end_ind_arr[tid]];
        if(start_ind_arr[tid] != -1)
        {
            if(tid == pivotIndex)
            {
                sh_gpu_suf_arr[pivotIndexPosition] = sh_gpu_suf_arr_copy[pivotIndex];
                start_ind_arr_copy[pivotIndexPosition] = -1;
                end_ind_arr_copy[pivotIndexPosition] = -1;
            }
            else if(tid == start_ind_arr[tid])
            {
                if(sh_gpu_aux_arr[tid] == 0)
                {
                    sh_gpu_suf_arr[pivotIndexPosition + tid - start_ind_arr[tid] - sh_gpu_aux_arr[tid] + 1] = sh_gpu_suf_arr_copy[tid];
                    start_ind_arr_copy[pivotIndexPosition + tid - start_ind_arr[tid] - sh_gpu_aux_arr[tid] + 1] = pivotIndexPosition + 1;
                }
                else
                {
                    sh_gpu_suf_arr[start_ind_arr[tid] + sh_gpu_aux_arr[tid] - 1] = sh_gpu_suf_arr_copy[tid];
                    end_ind_arr_copy[start_ind_arr[tid] + sh_gpu_aux_arr[tid] - 1] = pivotIndexPosition - 1;
                }
            }
            else
            {
                if(sh_gpu_aux_arr[tid-1] == sh_gpu_aux_arr[tid])      // if equal it means current value of gpu suf arr is greater than pivot
                {
                    if(tid > pivotIndex)
                    {
                        sh_gpu_suf_arr[pivotIndexPosition + tid - start_ind_arr[tid] - sh_gpu_aux_arr[tid]] = sh_gpu_suf_arr_copy[tid];
                        start_ind_arr_copy[pivotIndexPosition + tid - start_ind_arr[tid] - sh_gpu_aux_arr[tid]] = pivotIndexPosition + 1;                  
                    }
                    else
                    {
                        sh_gpu_suf_arr[pivotIndexPosition + tid - start_ind_arr[tid] - sh_gpu_aux_arr[tid] + 1] = sh_gpu_suf_arr_copy[tid];
                        start_ind_arr_copy[pivotIndexPosition + tid - start_ind_arr[tid] - sh_gpu_aux_arr[tid] + 1] = pivotIndexPosition + 1;
                    }
                }
                else
                {
                    sh_gpu_suf_arr[start_ind_arr[tid] + sh_gpu_aux_arr[tid] - 1] = sh_gpu_suf_arr_copy[tid];
                    end_ind_arr_copy[start_ind_arr[tid] + sh_gpu_aux_arr[tid] - 1] = pivotIndexPosition - 1;
                }
            }
            terminate_t = 1;
        }
        else
        {
            sh_gpu_suf_arr[tid] = sh_gpu_suf_arr_copy[tid];
        }
    }
}

__global__ void adjust_prefix_sum(int *sh_gpu_aux_arr, int *sh_gpu_aux_arr_copy, int *start_ind_arr, int *end_ind_arr, int n)
{
    int tid = (int) (threadIdx.x + blockDim.x * threadIdx.y + \
                            ( blockIdx.x * blockDim.x * blockDim.y ) \
                            + ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x));                 // This gives every thread a unique ID.

    if(tid < n)
    {
        if(start_ind_arr[tid] != 0)
        {
                sh_gpu_aux_arr_copy[tid] = sh_gpu_aux_arr[tid] - sh_gpu_aux_arr[start_ind_arr[tid] - 1];
        }
        else
        {
            sh_gpu_aux_arr_copy[tid] = sh_gpu_aux_arr[tid];
        }
    }
}

__global__ void init_kernel(int *start_ind_arr, int *end_ind_arr, int *start_ind_arr_copy, int *end_ind_arr_copy, int n)
{

    unsigned long int tid = (unsigned long int) (threadIdx.x + blockDim.x * threadIdx.y + \
                            ( blockIdx.x * blockDim.x * blockDim.y ) \
                            + ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x));                 // This gives every thread a unique ID.

    if(tid < n)
    {
        start_ind_arr[tid] = 0;
        start_ind_arr_copy[tid] = 0;
        end_ind_arr[tid] = n-1;
        end_ind_arr_copy[tid] = n-1;
    }

}

__global__ void init_suffix_array(int *device_arr, int n)
{

    unsigned long int tid = (unsigned long int) (threadIdx.x + blockDim.x * threadIdx.y + \
                            ( blockIdx.x * blockDim.x * blockDim.y ) \
                            + ( blockIdx.y * blockDim.x * blockDim.y * gridDim.x));                 // This gives every thread a unique ID.

    if(tid < n)
    {
        device_arr[tid] = tid;
    }

}

void quick_sort_genome(int *device_arr, long unsigned int suff_size)
{

    sh_gpu_suf_arr = device_arr;
    
    init_kernel <<<blockGridRows, threadBlockRows>>> (start_ind_arr, end_ind_arr, start_ind_arr_copy, end_ind_arr_copy, suff_size);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Kernel Execution
    int end_loop = 0;
    while(true)
    {   
        int total_shared_memory = block_size*(sizeof(int));
        quickSortGPU <<<blockGridRows, 
                        threadBlockRows, 
                        total_shared_memory>>> (sh_gpu_suf_arr, sh_gpu_suf_arr_copy, 
                                                sh_gpu_aux_arr, start_ind_arr, 
                                                end_ind_arr, start_ind_arr_copy, 
                                                end_ind_arr_copy, gpu_genome, 
                                                block_size, suff_size);

        //CUT_CHECK_ERROR("Quick sort Kernel execution failed\n");
        CUDA_SAFE_CALL( hipDeviceSynchronize() );
        
        prefixCompute(sh_gpu_aux_arr, blockGridRows, threadBlockRows, block_size, 0, suff_size-1, suff_size);
    
        adjust_prefix_sum <<< blockGridRows, threadBlockRows >>> (sh_gpu_aux_arr, sh_gpu_aux_arr_copy, start_ind_arr, end_ind_arr, suff_size);
        CUDA_SAFE_CALL( hipDeviceSynchronize() );

        write_pivot <<< blockGridRows, threadBlockRows >>> (sh_gpu_suf_arr, sh_gpu_suf_arr_copy, sh_gpu_aux_arr_copy, start_ind_arr, end_ind_arr, start_ind_arr_copy, end_ind_arr_copy, suff_size);
        CUDA_SAFE_CALL( hipDeviceSynchronize() );

        hipMemcpyFromSymbol(&end_loop, HIP_SYMBOL(terminate_t), sizeof(end_loop), 0, hipMemcpyDeviceToHost);

//        printf("Execute More:- %d \n", end_loop);
        if(end_loop == 0)
        {
            break;
        }
    }   

}

void set_quickSort_kernel(int suff_size)
{
    // Get Device Properties
    hipDeviceProp_t prop;
    int count; 
//    int MAX_THREADS_PER_BLOCK;
    int maxBlockGridWidth, maxBlockGridHeight, maxBlockGridDepth;
    hipGetDeviceCount(&count);
    hipGetDeviceProperties(&prop, 0);
//    MAX_THREADS_PER_BLOCK = prop.maxThreadsPerBlock;
    maxBlockGridWidth = prop.maxGridSize[0];
    maxBlockGridHeight = prop.maxGridSize[1];
    maxBlockGridDepth = prop.maxGridSize[2];

    // Block and grid dimensions
    block_size = 512; // MAX_THREADS_PER_BLOCK;

    if((unsigned long int)suff_size > (int)(maxBlockGridWidth * maxBlockGridHeight * maxBlockGridDepth * block_size))
    {
        cout << "Suffix Array Length out of Device block Size" << endl;
        exit(1);        
    }

    int blockGridWidth = suff_size/block_size + 1;
    int blockGridHeight = 1;
    if(blockGridWidth > maxBlockGridWidth)
    {
        blockGridWidth = maxBlockGridWidth;     
        blockGridHeight = (suff_size/(maxBlockGridWidth * block_size)) + 1;
    }

    blockGridRows.x = blockGridWidth; 
    blockGridRows.y = blockGridHeight;
    threadBlockRows.x = block_size;
    threadBlockRows.y = 1;
}

void alloc_device_pointers(unsigned long int suff_size)
{
    // Allocating memory    
    CUDA_SAFE_CALL( hipMalloc( (void **)&sh_gpu_suf_arr_copy, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&start_ind_arr_copy, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&end_ind_arr_copy, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&start_ind_arr, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&end_ind_arr, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&sh_gpu_aux_arr, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&sh_gpu_aux_arr_copy, sizeof(int) * suff_size) );

}

void free_gpu_memory(){
    hipFree(sh_gpu_suf_arr_copy);
    hipFree(sh_gpu_aux_arr);
    hipFree(sh_gpu_aux_arr_copy);
    hipFree(start_ind_arr);
    hipFree(end_ind_arr);
    hipFree(start_ind_arr_copy);
    hipFree(end_ind_arr_copy);
}

void free_memory()
{
    free(cpu_genome);
    free(cpu_suf_arr);
    free(cpu_final_arr);
    hipFree(gpu_genome);
    hipFree(gpu_aux_arr);
    hipFree(gpu_suf_arr);
}

static int max_bucket_size = -1;

void quick_sort_bucket(int *device_arr, int bucket_size, int bucket_number, bool last_bucket){

    if(bucket_number == 0){
        alloc_device_pointers(bucket_size);
        max_bucket_size = bucket_size;        
    } else if(max_bucket_size < bucket_size){
        free_gpu_memory();
        alloc_device_pointers(bucket_size);
        max_bucket_size = bucket_size;
    }

    quick_sort_genome(device_arr, bucket_size);

    if(last_bucket){
        free_gpu_memory();
    }

}

void init_arr(int suff_size)
{
    for(int i=0;i<suff_size;i++)
    {
        cpu_suf_arr[i] = (int)i;
        //printf("%d - %s\n", cpu_suf_arr[i], cpu_genome+i);
    }
    for(int i=0;i<suff_size;i++)
    {
        cpu_final_arr[i] = 0;
    } 
}

void copy_to_memory( int suff_size)
{
    // Copy the data to the device
    hipMemcpy( gpu_suf_arr, cpu_suf_arr, sizeof(int) * suff_size, 
                hipMemcpyHostToDevice);
    
    hipMemcpy( gpu_genome, cpu_genome, sizeof(char) * suff_size, 
                hipMemcpyHostToDevice);
    
    hipMemcpy( gpu_aux_arr, cpu_final_arr, sizeof(int) * suff_size, 
                hipMemcpyHostToDevice);
}

int main( int argc, char** argv) 
{
    int suff_size = atoi(argv[1]);

    /* Read genome from disk */
    setup(suff_size, argv[2] );

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    cpu_final_arr = (int*) malloc( sizeof(int) * suff_size);
    cpu_suf_arr = (int*) malloc( sizeof(int) * suff_size);
    CUDA_SAFE_CALL( hipMalloc( (void **)&gpu_aux_arr, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&gpu_suf_arr, sizeof(int) * suff_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&gpu_genome, sizeof(char) * suff_size) );

    set_quickSort_kernel(suff_size);

    init_arr(suff_size);
    copy_to_memory(suff_size);

    quick_sort_bucket(gpu_suf_arr, suff_size, 0, true);

    CUDA_SAFE_CALL( hipMemcpy(cpu_final_arr, gpu_suf_arr, sizeof(int) * suff_size, hipMemcpyDeviceToHost) );
    cout << "Suffix Array for Genome: " << endl;
    print_gene_array(cpu_final_arr, suff_size);


    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

    printf("%d %f\n", suff_size, elapsedTime * (0.001));
    free_memory();

    return 0;
}
